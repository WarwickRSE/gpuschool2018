// -*- mode: C -*-

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>  // clock() and clock_t
#include <hip/hip_runtime.h>

// Global data which defines f(x,y,z)

// Number of points P
#define P 500      

// Array of points at which Gaussians are located. This
// is now a static array in constant memory
//double *points;
__constant__ double points[3*P];
     
// Gaussian height parameter A
__constant__ double A = 0.1;

// Gaussian width parameter w
__constant__ double w = 0.2;

// This function evaluates f(x,y,z). It is only
// visible to the device and cannot be called
// directly by the host.
__device__ double func(double x, double y, double z) {

  double tot,arg;
  int i;
  
  tot = 0.0;
  for (i=0;i<P;i++) {
    arg  = 0.0;
    arg += (x-points[3*i+0])*(x-points[3*i+0]);
    arg += (y-points[3*i+1])*(y-points[3*i+1]);
    arg += (z-points[3*i+2])*(z-points[3*i+2]);
    tot += A*expf(-w*arg);
  }

  return tot;

}

// This is our kernel. for a point on our two-dimension grid
// integrate over the function in the third dimension.
// kernels must be global, visible to the host and device.
__global__ void integrate1D(double Min, double Max, int Ngrid, double *g_dev ) {

  int i,j,k;
  double delta,x,y,z,p_old,p_new,g_loc;

  // Which element on the 2D grid does this instance 
  // of the kernel need to compute?
  i = blockIdx.x*blockDim.x + threadIdx.x;
  j = blockIdx.y*blockDim.y + threadIdx.y;

  if ( (i<Ngrid) && (j <Ngrid) ) {

    // Compute grid spacing and current x and y value
    delta = (Max-Min)/(double)(Ngrid-1);
    x     = Min+(double)i*delta;
    y     = Min+(double)j*delta;
    
    // Integrate along z using trapezoidal rule
    g_loc = 0.0;
    z     = Min;
    p_old = expf(func(x,y,z));
    for (k=1;k<Ngrid;k++) {
      z += delta;
      p_new  = expf(func(x,y,z));
      g_loc += delta*0.5f*(p_old+p_new);
      p_old  = p_new;
    }
    
    // Store at the appropriate location in g
    g_dev[i*Ngrid + j] = g_loc;

  }

  return;

}

/*================================================
  This program uses the data and functions above 
  module above to compute the two-dimensional 
  function g(x,y). Note that we store the function 
  in a long vector of length Ngrid*Ngrid rather 
  than in a 2D matrix.
!===============================================*/
int main (int argc, char *argv[]) {

  // Extend of domain in each dimension
  double gridMax =  10.0;
  double gridMin = -10.0;

  // Number of grid points in each dimension
  int Ngrid = 128;

  // Array to hold set of P points on the host
  double *tmpPoints;

  // Memory for g on the host
  double *g_host;

  // Memory for g on the device
  double *g_dev;

  // Variables to hold the dimensions of the block
  // and thread grids. The dim3 type is provided in cuda.h
  int blocksPerGrid,threadsPerBlock;
  dim3 blocks,threads;

  // cudaError_t is a type defined in cuda.h
  hipError_t err;

  int i,j,count,idev;
  double x;

  // Process command line argument
  if (argc != 2) {
    printf("Usage : quad_gpu <device_id>\n");
    exit(EXIT_FAILURE);
  }

  int in_dev = atoi(argv[1]);

  // Make sure we have a CUDA capable device to work with
  err = hipGetDeviceCount(&count);
  if ( (count==0) || (err!=hipSuccess) ) {
    printf("No CUDA supported devices are available in this system.\n");
    exit(EXIT_FAILURE);
  } else {
    printf("Found %d CUDA devices in this system\n",count);
  }

  err = hipSetDevice(in_dev);
  if ( err!=hipSuccess ) {
    printf("Error setting active device\n");
    exit(EXIT_FAILURE);
  }

  err = hipGetDevice(&idev);
  if ( err!=hipSuccess ) {
    printf("Error identifying active device\n");
    exit(EXIT_FAILURE);
  }

  printf("Using device %d\n",idev);

  // Allocate memory for points on the host
  tmpPoints = (double *)malloc(3*P*sizeof(double));

  // Populate tmpPoints with random numbers between -10 and 10
  for (i=0;i<P;i++) {
    for (j=0;j<3;j++) {
      x = rand()/(double)RAND_MAX;
      tmpPoints[3*i+j] = gridMin + (gridMax-gridMin)*x;
    }
  }

  // Allocate memory for points on the device
  //err = cudaMalloc(&points,3*P*sizeof(double));
  //if ( err!=cudaSuccess ) {
  //  printf("Error allocating memory for points on device\n");
  //  exit(EXIT_FAILURE);
  //}
  
  
  // Copy from tmpPoints on the host to points on the device
  //err = cudaMemcpy(points,tmpPoints,3*P*sizeof(double),cudaMemcpyHostToDevice);
  err = hipMemcpyToSymbol(HIP_SYMBOL(points),tmpPoints,3*P*sizeof(double));
  if ( err!=hipSuccess ) {
    printf("Error copying points to device\n");
    exit(EXIT_FAILURE);
  }

  printf("Copied array of points to device memory\n");

  // Release memory on the host
  free(tmpPoints);

  // Allocate memory for g on the device and zero it out
  err = hipMalloc(&g_dev,Ngrid*Ngrid*sizeof(double));
  if ( err!=hipSuccess ) {
    printf("Error allocating memory for g_dev on device\n");
    exit(EXIT_FAILURE);
  }
  hipMemset(g_dev,0,Ngrid*Ngrid*sizeof(double));

  // Allocate memory for g on the host and zero it out
  g_host = (double *)malloc(Ngrid*Ngrid*sizeof(double));
  memset(g_host,0,Ngrid*Ngrid*sizeof(double));

  // We want a thread running the integrate1D kernel for every
  // point in g_dev that we want to evaluate. 
  // Pick a sensible block size
  blocksPerGrid   = 4;
 
  // Calculate the number of threads per block to make up the
  // entire grid of Ngrid*Ngrid threads
  threadsPerBlock = Ngrid/blocksPerGrid;
  if (Ngrid%blocksPerGrid!=0) { threadsPerBlock += 1; }

  // Multidimensional grid dimensions, use the dim3 type
  blocks.x  = blocksPerGrid   ; blocks.y  = blocksPerGrid   ; blocks.z  = 1;
  threads.x = threadsPerBlock ; threads.y = threadsPerBlock ; threads.z = 1;

  // Launch our kernel to compute g_dev on the device
  printf("Using block grid dimensions of %d by %d\n",blocks.x,blocks.y);
  printf("Thread grid within a block is  %d by %d\n",threads.x,threads.y);

  clock_t t1 = clock();

  printf("Launching %d threads\n",blocks.x*blocks.y*threads.x*threads.y);

  integrate1D<<<blocks,threads>>>(gridMin,gridMax,Ngrid,g_dev);
  hipDeviceSynchronize();

  clock_t t2 = clock();

  // Copy from the device to the host
  err = hipMemcpy(g_host,g_dev,Ngrid*Ngrid*sizeof(double),hipMemcpyDeviceToHost);
  if ( err != hipSuccess ) {
    printf("Error copying g from device to host\n");
    exit(EXIT_FAILURE);
  }

  printf("Time taken on GPU = %f milliseconds\n",(double)(t2-t1)*1000.0/(double)CLOCKS_PER_SEC);


  // Release device memory
  hipFree(g_dev);
  hipFree(points);

  // Release host memory
  free(g_host);

  return 0;

}
